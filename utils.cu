#include "hip/hip_runtime.h"
#include "utils.h"

//    template <typename ValueType> 
//    void rotationplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
void rotationplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
    {
      ValueType temp = cs * dx + sn * dy;
      dy = -sn*dx+cs*dy;
      dx = temp;
    }

//    template <typename ValueType>
//    void genererrotaionplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
void genererrotaionplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
    {
      if(dy == ValueType(0.0)){
			cs = 1.0;
			sn = 0.0;
      }else if (abs(dy) > abs(dx)) {
			ValueType tmp = dx / dy;
			sn = ValueType(1.0) / sqrt(ValueType(1.0) + tmp*tmp);
			cs = tmp*sn;            
      }else {
			ValueType tmp = dy / dx;
			cs = ValueType(1.0) / sqrt(ValueType(1.0) + tmp*tmp);
			sn = tmp*cs;
      }
    }

//    template <class LinearOperator,typename ValueType> 
//    void applyrotationplan(cusp::csr_matrix<int, ValueType, MemorySpace>& H, ValueType& cs, ValueType& sn, ValueType& s, int i)
void applyrotationplan(cusp::array2d<ValueType, LocalSpace, cusp::column_major>& H, CuspArray& cs, CuspArray& sn, CuspArray& s, int i)
    {
      for (int k = 0; k < i; k++){
			rotationplan(H(k,i), H(k+1,i), cs[k], sn[k]);
      }
      genererrotaionplan(H(i,i), H(i+1,i), cs[i], sn[i]);
      rotationplan(H(i,i), H(i+1,i), cs[i], sn[i]);
      rotationplan(s[i], s[i+1], cs[i], sn[i]);
    }
    
    
    

// cusp gmres modified. it runs on one gpu 
// coming a version running on multiple gpu(s) i guess

int my_GMRES(CudaMatrix& A, CudaVector& x,  CudaVector& b, int restart, cusp::default_monitor<ValueType>& monitor)
//	       Preconditioner& M)
{
  //    typedef typename LinearOperator::value_type   ValueType;
  //    typedef typename LinearOperator::memory_space MemorySpace;
 //     typedef typename norm_type<ValueType>::type NormType;
      // here we check that it's a squar matrix
      assert(A.num_rows == A.num_cols);        // sanity check
//      std::cout << "test aasert passé \n ";
      const size_t N = A.num_rows;
//      std::cout << "1111\n";
      const int R = restart;
      int i, j, k;
//      std::cout << " 2222\n";
      ValueType beta = 0;
      ValueType resid0 = 0;
      cusp::array1d<ValueType,cusp::host_memory> rel_resid(1);
//      std::cout << " 3333\n ";
      //allocate workspace
      cusp::array1d<ValueType,MemorySpace> w(N);
//           std::cout << "3bis\n";
      cusp::array1d<ValueType,MemorySpace> V0(N); //Arnoldi matrix pos 0
              std::cout << "3bisbis N =" << N << " R " << R << "\n";
      cusp::array2d<ValueType,cusp::device_memory,cusp::column_major> V(N,R+1,ValueType(0.0)); //Arnoldi matrix
      
 		     std::cout << "4444\n";
      //duplicate copy of s on GPU
      cusp::array1d<ValueType,MemorySpace> sDev(R+1);
      std::cout << " 5555 \n";
      //HOST WORKSPACE
      cusp::array2d<ValueType,cusp::host_memory,cusp::column_major> H(R+1, R); //Hessenberg matrix
      cusp::array1d<ValueType,cusp::host_memory> s(R+1);
      cusp::array1d<ValueType,cusp::host_memory> cs(R);
      cusp::array1d<ValueType,cusp::host_memory> sn(R);
      std::cout << " 66666\n";
      ValueType b_norm = blas::nrm2(b);
      std::cout << " 77777\n";
      do{
      	std::cout << "on entre dans la boucle principale \n";
			// compute initial residual and its norm //
			cusp::multiply(A, x, w);                     // V(0) = A*x        //
			blas::axpy(b,w,ValueType(-1));               // V(0) = V(0) - b   //
		//	cusp::multiply(M,w,w);                       // V(0) = M*V(0)     //
			beta = blas::nrm2(w);                        // beta = norm(V(0)) //
			blas::scal(w, ValueType(-1.0/beta));         // V(0) = -V(0)/beta //
			blas::copy(w,V.column(0));
			// save very first residual norm //
			if (monitor.iteration_count()== 0){
			  //resid0 = beta;
		//	  cusp::multiply(M,b,V0);
			  resid0 = blas::nrm2(V0)/b_norm;
			}
			std::cout << "premier test reussi \n";
			//s = 0 //
			blas::fill(s,ValueType(0.0));
			s[0] = beta;
			i = -1;
	
			do{
				std::cout << "on entre dans la seconde boucle do \n";
			  ++i;
			  ++monitor;
			  
			  //apply preconditioner
			  //can't pass in ref to column in V so need to use copy (w)
			  cusp::multiply(A,w,V0);
			  //V(i+1) = A*w = M*A*V(i)    //
		//	  cusp::multiply(M,V0,w);
			  
			  for (k = 0; k <= i; k++){
				 //  H(k,i) = <V(i+1),V(k)>    //
				 H(k, i) = blas::dotc(w, V.column(k));
				 // V(i+1) -= H(k, i) * V(k)  //
				 blas::axpy(V.column(k),w,-H(k,i));
			  }
			  std::cout <<  "on sort d'une troisieme boucle\n"; 
			  H(i+1,i) = blas::nrm2(w);   
			  // V(i+1) = V(i+1) / H(i+1, i) //
			  blas::scal(w,ValueType(1.0)/H(i+1,i));
			  blas::copy(w,V.column(i+1));
			  std::cout << " avant la fameuse rotation \n";
			  applyrotationplan(H,cs,sn,s,i);
			  std::cout << " apres la fameuse rotation \n";
			  rel_resid[0] = abs(s[i+1]) / resid0 + monitor.absolute_tolerance();
			  
			  //check convergence condition
			  //if (rel_resid < monitor.relative_tolerance())
			  if (monitor.finished(rel_resid)){
				 break;
			  }
			}while (i+1 < R && monitor.iteration_count()+1 <= monitor.iteration_limit());
			std::cout << " on sort de la seconde boucle \n ";	

			// solve upper triangular system in place //
			for (j = i; j >= 0; j--){
			  s[j] /= H(j,j);
			  //S(0:j) = s(0:j) - s[j] H(0:j,j)
			  for (k = j-1; k >= 0; k--){
				 s[k] -= H(k,j) * s[j];
			  }
			}
			std::cout << "on sort d'une quatrieme boucle \n";
			// update the solution //
	
			//copy s to gpu 
			blas::copy(s,sDev);
			// x= V(1:N,0:i)*s(0:i)+x //
			for (j = 0; j <= i; j++){
			  // x = x + s[j] * V(j) //
			  blas::axpy(V.column(j),x,s[j]);
			}
			std::cout << " on sort d'une cinquieme boucle \n";
		} while (rel_resid[0] >= monitor.tolerance() &&  monitor.iteration_count()+1 <= monitor.iteration_limit());
	 	return 0;
}
    
 
