#include "hip/hip_runtime.h"
#include "utils.h"

//    template <typename ValueType> 
    void rotationplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
    {
      ValueType temp = cs * dx + sn *dy;
      dy = -sn*dx+cs*dy;
      dx = temp;
    }

//    template <typename ValueType>
    void genererrotaionplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
    {
      if(dy == ValueType(0.0)){
			cs = 1.0;
			sn = 0.0;
      }else if (abs(dy) > abs(dx)) {
			ValueType tmp = dx / dy;
			sn = ValueType(1.0) / sqrt(ValueType(1.0) + tmp*tmp);
			cs = tmp*sn;            
      }else {
			ValueType tmp = dy / dx;
			cs = ValueType(1.0) / sqrt(ValueType(1.0) + tmp*tmp);
			sn = tmp*cs;
      }
    }

//    template <class LinearOperator,typename ValueType> 
    void applyrotationplan(LinearOperator& H, ValueType& cs, ValueType& sn, ValueType& s, int i)
    {
      for (int k = 0; k < i; k++){
			rotationplan(H(k,i), H(k+1,i), cs[k], sn[k]);
      }
      GeneratePlaneRotation(H(i,i), H(i+1,i), cs[i], sn[i]);
      rotationplan(H(i,i), H(i+1,i), cs[i], sn[i]);
      rotationplan(s[i], s[i+1], cs[i], sn[i]);
    }
