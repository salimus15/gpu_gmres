#include "hip/hip_runtime.h"
#include "utils.h"

//    template <typename ValueType> 
//    void rotationplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
void rotationplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
    {
      ValueType temp = cs * dx + sn * dy;
      dy = -sn*dx+cs*dy;
      dx = temp;
    }

//    template <typename ValueType>
//    void genererrotaionplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
void genererrotaionplan(ValueType& dx, ValueType& dy, ValueType& cs, ValueType& sn)
    {
      if(dy == ValueType(0.0)){
			cs = 1.0;
			sn = 0.0;
      }else if (abs(dy) > abs(dx)) {
			ValueType tmp = dx / dy;
			sn = ValueType(1.0) / sqrt(ValueType(1.0) + tmp*tmp);
			cs = tmp*sn;            
      }else {
			ValueType tmp = dy / dx;
			cs = ValueType(1.0) / sqrt(ValueType(1.0) + tmp*tmp);
			sn = tmp*cs;
      }
    }

//    template <class LinearOperator,typename ValueType> 
//    void applyrotationplan(cusp::csr_matrix<int, ValueType, MemorySpace>& H, ValueType& cs, ValueType& sn, ValueType& s, int i)
void applyrotationplan(cusp::array2d<ValueType, LocalSpace, cusp::column_major>& H, CuspArray& cs, CuspArray& sn, CuspArray& s, int i)
    {
      for (int k = 0; k < i; k++){
			rotationplan(H(k,i), H(k+1,i), cs[k], sn[k]);
      }
      genererrotaionplan(H(i,i), H(i+1,i), cs[i], sn[i]);
      rotationplan(H(i,i), H(i+1,i), cs[i], sn[i]);
      rotationplan(s[i], s[i+1], cs[i], sn[i]);
    }
