
#include <iostream>
#include "gmres_cuda.h"

using namespace std;

int main(int argc, char ** argv){
        int i;
	std::string filename;
	int tolerance, mGmres;
	
	if(argc > 10){
		printf("\nje sais pas trop !!!!");
		return 1;
	}
		
	for(i = 0; i < argc; ++i){
		// we check if the matrix is contained in a matrix market file 
		if (strcmp(argv[i], " --matrix-from-file") == 0){
			// we get the name of the file from where to get the matrix 
			filename.assign("./rdb968.mtx");
		}
	
		// we check if tolerance was specified 
		if (strcmp(argv[i], " --tolerance") == 0){
			// we get the value of the tolerance 
			tolerance = atoi(argv[i+1]);
		}
	
		// we check if number of iterations was specified
		if (strcmp(argv[i], " --restart") == 0){
			// we get the number of iterations before a restart 
			mGmres = atoi(argv[i+1]);
		}
	}

        if(cusp_GMRES(filename, tolerance, mGmres)== 0){
          cout << " it seems to be ok\n";
        }



	return 0;
}


