#include "gmres_cuda.h"
extern "C" {

// reading a matrix from a matrix market file 
int read_Operator_A_mm(CudaMatrix& mtx, const std::string& filename){
	std::cout << " Going to make read of the matrix \n";
	cusp::io::read_matrix_market_file(mtx, "rdb968.mtx");
	std::cout << " Matrix reading done \n";
	return 0;
}


//
int initialize_problem(CudaMatrix& mtx, const std::string& filename, CudaVector& b, CudaVector& x, int& mGmres, int& tolerance){
	//cusp::csr_matrix<int, double, cusp::device_memory> A;
	// allocate storage for solution (x) and right hand side (b)
	//cusp::array1d<ValueType, MemorySpace> x(A.num_rows, ValueType(1));
	//cusp::array1d<ValueType, MemorySpace> b(A.num_rows);
	
	read_Operator_A_mm( mtx, filename);
	std::cout << " Matrix read and has : " << mtx.num_rows << "rows " << mtx.cols << "cols " << mtx.num_entries << " entries \n";
	// here we gonna set the vectors sizes
	x.resize(mtx.num_rows);
	b.resize(mtx.num_rows);	

	// set initial guess
	thrust::fill( x.begin(), x.end(), ValueType(1) );	
	std::cout << " vector x set to size of : " << x.size << "\n";
	thrust::fill( b.begin(), b.end(), ValueType(2) );
	std::cout << " vector b set to size of : " << b.size << "\n";
	// set stopping criteria:
	//  iteration_limit    = 100
	//  relative_tolerance = 1e-6
	//	cusp::verbose_monitor<ValueType> monitor(b, 100, 1e-6);
	//	int restart = 50;
	
//	on initialise le moniteur de convergence
	
	return 0;
}

// calling the GMRES function implemented in CUSP
int call_cusp_GMRES(CudaMatrix& A, CudaVector& x, CudaVector b, int restart){
	 // solve the linear system A * x = b with the GMRES
	 
    cusp::krylov::gmres(A, x, b,restart);

	return 0;
}



//
int cusp_GMRES(int argc, char ** argv){
	int i;
	std::string filename;
	int tolerance, mGmres;
	
	CudaMatrix mtx;
	CudaVector x,b;

	if(argc > 10){
		printf("\nje sais pas trop !!!!");
		return 1;
	}
		
	for(i = 0; i < argc; ++i){
		// we check if the matrix is contained in a matrix market file 
		if (strcmp(argv[i], " --matrix-from-file") == 0){
			// we get the name of the file from where to get the matrix 
			filename.assign("./rdb968.mtx");
		}
	
		// we check if tolerance was specified 
		if (strcmp(argv[i], " --tolerance") == 0){
			// we get the value of the tolerance 
			tolerance = atoi(argv[i+1]);
		}
	
		// we check if number of iterations was specified
		if (strcmp(argv[i], " --restart") == 0){
			// we get the number of iterations before a restart 
			mGmres = atoi(argv[i+1]);
		}
	}
	
	//read_Operator_A_mm(mtx, filename);
	initialize_problem(mtx, filename, b, x, mGmres, tolerance);
	std::cout << "problem initialization done !\n ";

	call_cusp_GMRES( mtx, x, b, mGmres);
	std::cout << " gmres solving done !!!\n";
	return 0;
}

}// for the extern 
