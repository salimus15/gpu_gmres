#include "hip/hip_runtime.h"
#include "gmres_cuda.h"
extern "C" {

// reading a matrix from a matrix market file 
int read_Operator_A_mm(CudaMatrix& mtx, const std::string& filename){
	std::cout << " Going to make read of the matrix \n";
	cusp::io::read_matrix_market_file(mtx, "rdb968.mtx");
	std::cout << " Matrix reading done \n";
	return 0;
}


//
int initialize_problem(CudaMatrix& mtx, const std::string& filename, CudaVector& b, CudaVector& x, int& mGmres, int& tolerance){
	//cusp::csr_matrix<int, double, cusp::device_memory> A;
	// allocate storage for solution (x) and right hand side (b)
	//cusp::array1d<ValueType, MemorySpace> x(A.num_rows, ValueType(1));
	//cusp::array1d<ValueType, MemorySpace> b(A.num_rows);
	
	read_Operator_A_mm( mtx, filename);
	std::cout << " Matrix read and has : " << mtx.num_rows << "rows " << mtx.num_cols << "cols " << mtx.num_entries << " entries \n";
	// here we gonna set the vectors sizes
	x.resize(mtx.num_rows);
	b.resize(mtx.num_rows);	

	// set initial guess
	thrust::fill( x.begin(), x.end(), ValueType(1) );	
	std::cout << " vector x set to size of : " << x.size() << "\n";
	thrust::fill( b.begin(), b.end(), ValueType(2) );
	std::cout << " vector b set to size of : " << b.size() << "\n";
	// set stopping criteria:
	//  iteration_limit    = 100
	//  relative_tolerance = 1e-6
	//	cusp::verbose_monitor<ValueType> monitor(b, 100, 1e-6);
	//	int restart = 50;
	
//	on initialise le moniteur de convergence
	
	return 0;
}

// calling the GMRES function implemented in CUSP
int call_cusp_GMRES(CudaMatrix& A, CudaVector& x, CudaVector b, int restart){
	 // solve the linear system A * x = b with the GMRES
	 
    cusp::krylov::gmres(A, x, b,restart);

	return 0;
}



//
int cusp_GMRES(int argc, char ** argv){
	int i;
	std::string filename;
	int tolerance, mGmres;
	
	CudaMatrix mtx;
	CudaVector x,b;

	if(argc > 10){
		printf("\nje sais pas trop !!!!");
		return 1;
	}
		
	for(i = 0; i < argc; ++i){
		// we check if the matrix is contained in a matrix market file 
		if (strcmp(argv[i], " --matrix-from-file") == 0){
			// we get the name of the file from where to get the matrix 
			filename.assign("./rdb968.mtx");
		}
	
		// we check if tolerance was specified 
		if (strcmp(argv[i], " --tolerance") == 0){
			// we get the value of the tolerance 
			tolerance = atoi(argv[i+1]);
		}
	
		// we check if number of iterations was specified
		if (strcmp(argv[i], " --restart") == 0){
			// we get the number of iterations before a restart 
			mGmres = atoi(argv[i+1]);
		}
	}
	
	//read_Operator_A_mm(mtx, filename);
	initialize_problem(mtx, filename, b, x, mGmres, tolerance);
	std::cout << "problem initialization done !\n ";
	std::cout << " now follow the data states before callin gmres :\n";
	std::cout << " Matrix read and has : " << mtx.num_rows << "rows " << mtx.num_cols << "cols " << mtx.num_entries << " entries \n";
	std::cout << " vector x set to size of : " << x.size() << "\n";
	std::cout << " vector b set to size of : " << b.size() << "\n";
	cusp::default_monitor<ValueType> monitor(b, 100, 1e-6);
//	call_cusp_GMRES( mtx, x, b, mGmres);
	my_GMRES( mtx, x, b, mGmres, monitor );
	std::cout << " gmres solving done !!!\n";
	return 0;
}


// cusp gmres modified. it runs on one gpu 
// coming a version running on multiple gpu(s) i guess

int my_GMRES(CudaMatrix& A, CudaVector& x,  CudaVector& b, int restart, cusp::default_monitor<ValueType>& monitor)
//	       Preconditioner& M)
{
//      typedef typename LinearOperator::value_type   ValueType;
//      typedef typename LinearOperator::memory_space MemorySpace;
 //     typedef typename norm_type<ValueType>::type NormType;
      // here we check that it's a squar matrix
      assert(A.num_rows == A.num_cols);        // sanity check
      const size_t N = A.num_rows;
      const int R = restart;
      int i, j, k;
      ValueType beta = 0;
      ValueType resid0 = 0;
      cusp::array1d<ValueType,cusp::host_memory> rel_resid(1);
      //allocate workspace
      cusp::array1d<ValueType,MemorySpace> w(N);
      cusp::array1d<ValueType,MemorySpace> V0(N); //Arnoldi matrix pos 0
      cusp::array2d<ValueType,MemorySpace,cusp::column_major> V(N,R+1,ValueType(0.0)); //Arnoldi matrix
      //duplicate copy of s on GPU
      cusp::array1d<ValueType,MemorySpace> sDev(R+1);
      //HOST WORKSPACE
      cusp::array2d<ValueType,cusp::host_memory,cusp::column_major> H(R+1, R); //Hessenberg matrix
      cusp::array1d<ValueType,cusp::host_memory> s(R+1);
      cusp::array1d<ValueType,cusp::host_memory> cs(R);
      cusp::array1d<ValueType,cusp::host_memory> sn(R);
      ValueType b_norm = blas::nrm2(b);
      
      do{
			// compute initial residual and its norm //
			cusp::multiply(A, x, w);                     // V(0) = A*x        //
			blas::axpy(b,w,ValueType(-1));               // V(0) = V(0) - b   //
		//	cusp::multiply(M,w,w);                       // V(0) = M*V(0)     //
			beta = blas::nrm2(w);                        // beta = norm(V(0)) //
			blas::scal(w, ValueType(-1.0/beta));         // V(0) = -V(0)/beta //
			blas::copy(w,V.column(0));
			// save very first residual norm //
			if (monitor.iteration_count()== 0){
			  //resid0 = beta;
		//	  cusp::multiply(M,b,V0);
			  resid0 = blas::nrm2(V0)/b_norm;
			}
			//s = 0 //
			blas::fill(s,ValueType(0.0));
			s[0] = beta;
			i = -1;
	
			do{
			  ++i;
			  ++monitor;
			  
			  //apply preconditioner
			  //can't pass in ref to column in V so need to use copy (w)
			  cusp::multiply(A,w,V0);
			  //V(i+1) = A*w = M*A*V(i)    //
		//	  cusp::multiply(M,V0,w);
			  
			  for (k = 0; k <= i; k++){
				 //  H(k,i) = <V(i+1),V(k)>    //
				 H(k, i) = blas::dotc(w, V.column(k));
				 // V(i+1) -= H(k, i) * V(k)  //
				 blas::axpy(V.column(k),w,-H(k,i));
			  }
			  
			  H(i+1,i) = blas::nrm2(w);   
			  // V(i+1) = V(i+1) / H(i+1, i) //
			  blas::scal(w,ValueType(1.0)/H(i+1,i));
			  blas::copy(w,V.column(i+1));
			  
			  applyrotationplan(H,cs,sn,s,i);
			  
			  rel_resid[0] = abs(s[i+1]) / resid0 + monitor.absolute_tolerance();
			  
			  //check convergence condition
			  //if (rel_resid < monitor.relative_tolerance())
			  if (monitor.finished(rel_resid)){
				 break;
			  }
			}while (i+1 < R && monitor.iteration_count()+1 <= monitor.iteration_limit());
	

			// solve upper triangular system in place //
			for (j = i; j >= 0; j--){
			  s[j] /= H(j,j);
			  //S(0:j) = s(0:j) - s[j] H(0:j,j)
			  for (k = j-1; k >= 0; k--){
				 s[k] -= H(k,j) * s[j];
			  }
			}
	
			// update the solution //
	
			//copy s to gpu 
			blas::copy(s,sDev);
			// x= V(1:N,0:i)*s(0:i)+x //
			for (j = 0; j <= i; j++){
			  // x = x + s[j] * V(j) //
			  blas::axpy(V.column(j),x,s[j]);
			}
		} while (rel_resid[0] >= monitor.tolerance() &&  monitor.iteration_count()+1 <= monitor.iteration_limit());
	 	return 0;
}

}// for the extern 
