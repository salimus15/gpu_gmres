#include "gmres_cuda.h"


// reading a matrix from a matrix market file 
int read_Operator_A_mm(CudaMatrix& mtx, const std::string& filename){

	cusp::io::read_matrix_market_file(mtx, filename);
	return 0;
}


//
int initialize_problem(CudaMatrix& mtx, const std::string& filename, CudaVector& b, CudaVector& x, cusp::default_monitor<ValueType>& monitor, int& mGmres, int& tolerance){
	//cusp::csr_matrix<int, double, cusp::device_memory> A;
	// allocate storage for solution (x) and right hand side (b)
	//cusp::array1d<ValueType, MemorySpace> x(A.num_rows, ValueType(1));
	//cusp::array1d<ValueType, MemorySpace> b(A.num_rows);
	
	read_Operator_A_mm( mtx, filename);
	// set initial guess
   thrust::fill( x.begin(), x.end(), ValueType(1) );	
	// set stopping criteria:
	//  iteration_limit    = 100
	//  relative_tolerance = 1e-6
	//	cusp::verbose_monitor<ValueType> monitor(b, 100, 1e-6);
	//	int restart = 50;
	
//	on initialise le moniteur de convergence
	monitor(b, mGmres, tolerance);
	return 0;
}

// calling the GMRES function implemented in CUSP
int call_cusp_GMRES(CudaMatrix& A, CudaVector& x, CudaVector b, int restart, cusp::default_monitor<ValueType>& monitor){
	 // solve the linear system A * x = b with the GMRES
    cusp::krylov::gmres(A, x, b,restart, monitor);

	return 0;
}



//
int cusp_GMRES(int argc, char ** argv){
	int i;
	char * filename;
	int tolerance, mGmres;
	cusp::default_monitor<ValueType> monitor;
	CudaMatrix mtx;
	CudaVector x,b;

	if(argc < 10){
		printf("\nje sais pas trop !!!!");
		return 1;
	}
		
	for(i = 0; i < argc; ++i){
		// we check if the matrix is contained in a matrix market file 
		if (strcmp(argv[i], " --matrix-from-file") == 0){
			// we get the name of the file from where to get the matrix 
			filename = argv[i+1];
		}
	
		// we check if tolerance was specified 
		if (strcmp(argv[i], " --tolerance") == 0){
			// we get the value of the tolerance 
			tolerance = atoi(argv[i+1]);
		}
	
		// we check if number of iterations was specified
		if (strcmp(argv[i], " --restart") == 0){
			// we get the number of iterations before a restart 
			mGmres = atoi(argv[i+1]);
		}
	}
	
	read_Operator_A_mm(mtx, filename);
	initialize_problem(mtx, filename, b, x, monitor, mGmres, tolerance);
	call_cusp_GMRES( mtx, x, b, mGmres, monitor);
	return 0;
}

